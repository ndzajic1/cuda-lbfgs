#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <iostream>
#include <vector>
#include <functional>

#include <cmath>
#include <chrono>
#include <algorithm> 
#include <random> 

using namespace std;


// Armijo and Wolfe condition constants
constexpr double C1 = 1e-4;   // Armijo condition constant
constexpr double C2 = 0.7;    // Wolfe condition constant

// Initial step size for line search
constexpr double INITIAL_STEP_SIZE = 1.0;

// Backtracking parameters
constexpr double BACKTRACKING_ALPHA = 0.5;  // Step reduction factor
constexpr double BACKTRACKING_TOL = 1e-8;  // Convergence threshold

// Wolfe interpolation parameters
constexpr double WOLFE_INTERP_MIN = 1e-10;
constexpr double WOLFE_INTERP_MAX = 10.0;


constexpr double MAX_STEP_SIZE = 10.0;
constexpr double MIN_STEP_SIZE = 1e-6;



// CUDA kernel for updating s and y
// CUDA kernel for sk ← xk+1 − xk. && CUDA kernel for yk ← gk+1 − gk.
__global__ void updateVectors(
    double* s, const double* x_new, const double* x_old,
    double* y, const double* g_new, const double* g_old,
    int size
) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        s[idx] = x_new[idx] - x_old[idx];
        y[idx] = g_new[idx] - g_old[idx];
    }
}


__global__ void copyGradient(double* q, const double* g, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        q[idx] = g[idx];
    }
}


// CUDA kernel for dk ← −r.
__global__ void negateVector(const double* r, double* d, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        d[idx] = -r[idx];
    }
}

// CUDA kernel for xk+1 ← xk + αkdk.
__global__ void updateSolution(const double* x_k, const double* d_k, double* x_next, double alpha, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        x_next[idx] = x_k[idx] + alpha * d_k[idx];
    }
}

__global__ void scaleByRho(const double* input, double* output, double rho, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size) {
        output[idx] = input[idx] * rho;
    }
}






// Utility functions for error handling
void checkCudaError(hipError_t err, const char* msg) {
    if (err != hipSuccess) {
        cerr << msg << " (" << hipGetErrorString(err) << ")" << endl;
        exit(EXIT_FAILURE);
    }
}

void checkCublasError(hipblasStatus_t err, const char* msg) {
    if (err != HIPBLAS_STATUS_SUCCESS) {
        cerr << msg << " (" << (err == HIPBLAS_STATUS_NOT_INITIALIZED ? "Not initialized" : "Other error") << ")" << endl;
        exit(EXIT_FAILURE);
    }
}



// Helper functions from sequential code
void ensureSameSize(const vector<double> &v1, const vector<double> &v2) {
    if (v1.size() != v2.size())
        throw logic_error("Vectors must be of same size");
}

double dotProduct(const vector<double> &v1, const vector<double> &v2) {
    ensureSameSize(v1, v2);

    double sum = 0.;
    for (int i = 0; i < v1.size(); ++i) 
        sum += v1[i] * v2[i];

    return sum;
}

vector<double> scalarProduct(const double scalar, const vector<double> &v) {
    vector<double> result = v;

    for (double &element : result)
        element *= scalar;

    return result;
}

vector<double> add(const vector<double> &v1, const vector<double> &v2) {
    ensureSameSize(v1, v2);

    vector<double> sum(v1.size());

    for (int i = 0; i < v1.size(); ++i)
        sum[i] = v1[i] + v2[i];

    return sum;
}



// Helper functions for interpolation
double cubicInterpolate(double alpha0, double alpha1, double phi0, double dphi0, double phi1, double dphi1) {
    double d1 = dphi0 + dphi1 - 3*(phi1 - phi0)/(alpha1 - alpha0);
    double d2 = std::copysign(sqrt(d1*d1 - dphi0*dphi1), alpha1 - alpha0);
    return alpha0 + (alpha1 - alpha0)*(dphi0 + d2 - d1)/(dphi0 - dphi1 + 2*d2);
}

double quadraticInterpolate(double alpha0, double alpha1, double phi0, double dphi0, double phi1) {
    return alpha0 - 0.5*dphi0*alpha0*alpha0/(phi1 - phi0 - dphi0*alpha0);
}

// Backtracking line search (Armijo condition)
double backtrackingLineSearch(const vector<double>& x, const vector<double>& d,
                              const function<double(vector<double>)>& f, 
                              const vector<double>& gradient) {
    double alpha = INITIAL_STEP_SIZE;
    
    while (f(x) - f(add(x, scalarProduct(alpha, d))) < C1 * alpha * dotProduct(gradient, d)) {
        alpha *= BACKTRACKING_ALPHA;
        if (alpha < BACKTRACKING_TOL) break;
    }
    
    if (alpha < 1e-4) {  
        return 0.5;     // Prevent excessively small steps
    }
    return alpha;
}

double backtrackingWolfeLineSearch(
    const std::vector<double>& x, 
    const std::vector<double>& d,
    const std::function<double(std::vector<double>)>& f, 
    const std::function<std::vector<double>(std::vector<double>)>& grad, 
    const std::vector<double>& gradient) {
    
    const double C1 = 1e-4;
    const double C2 = 0.9;
    const double INITIAL_STEP_SIZE = 1.0;
    const double BACKTRACKING_ALPHA = 0.5;
    const double BACKTRACKING_TOL = 1e-10;
    
    double alpha = INITIAL_STEP_SIZE;
    int iter = 0;
    
    double f_current = f(x);
    double gradient_dot_d = 0.0;
    for (size_t i = 0; i < x.size(); i++) {
        gradient_dot_d += gradient[i] * d[i];
    }
    
    // Cache previously computed points to avoid redundant calculations
    std::unordered_map<double, std::pair<double, std::vector<double>>> cache;
    
    double alpha_lo = 0.0;
    double alpha_hi = std::numeric_limits<double>::max();
    
    while (iter++ < 20) {
        if (cache.find(alpha) != cache.end()) {
            auto cached_result = cache[alpha];
            double f_new = cached_result.first;
            
            if (f_new <= f_current + C1 * alpha * gradient_dot_d) {
                std::vector<double> gradient_new = grad(cached_result.second);
                double gradient_new_dot_d = 0.0;
                for (size_t i = 0; i < x.size(); i++) {
                    gradient_new_dot_d += gradient_new[i] * d[i];
                }
                
                if (gradient_new_dot_d >= C2 * gradient_dot_d) {
                    break; 
                } else {
                    alpha_lo = alpha; 
                }
            } else {
                alpha_hi = alpha; 
            }
        } else {
            std::vector<double> x_new(x.size());
            for (size_t i = 0; i < x.size(); i++) {
                x_new[i] = x[i] + alpha * d[i];
            }
            
            double f_new = f(x_new);
            cache[alpha] = std::make_pair(f_new, x_new);
            
            if (f_new <= f_current + C1 * alpha * gradient_dot_d) {
                std::vector<double> gradient_new = grad(x_new);
                double gradient_new_dot_d = 0.0;
                for (size_t i = 0; i < x.size(); i++) {
                    gradient_new_dot_d += gradient_new[i] * d[i];
                }
                
                if (gradient_new_dot_d >= C2 * gradient_dot_d) {
                    break; 
                } else {
                    alpha_lo = alpha; 
                }
            } else {
                alpha_hi = alpha; 
            }
        }
        
        if (alpha_hi < std::numeric_limits<double>::max()) {
            alpha = (alpha_lo + alpha_hi) / 2.0;
        } else {
            alpha = 2.0 * alpha_lo;
        }
        
        if (alpha < BACKTRACKING_TOL) break;
    }
    
    return alpha;
}


double armijoInterpolationLineSearch(
    const vector<double>& x,
    const vector<double>& d,
    const function<double(vector<double>)>& f,
    const vector<double>& gradient
) {
    const double f_x = f(x);
    const double grad_dot_d = dotProduct(gradient, d);
    double alpha = INITIAL_STEP_SIZE;
    const int N = x.size();
    vector<double> x_new(N);
    double alpha_prev = 0.0;
    double f_prev = f_x;
    
    int iteration = 0;
    const int max_iterations = 20;  // Prevent infinite loops
    
    while (iteration++ < max_iterations) {
        for (int i = 0; i < N; ++i) {
            x_new[i] = x[i] + alpha * d[i];
        }
        
        double f_new = f(x_new);
        
        if (f_new <= f_x + C1 * alpha * grad_dot_d) {
            return alpha;
        }
        
        if (alpha < WOLFE_INTERP_MIN) {
            return WOLFE_INTERP_MIN;
        }
        
        if (alpha_prev > 0) {
            double delta_alpha = alpha - alpha_prev;
            if (abs(delta_alpha) < 1e-10) {
                alpha *= 0.5;  
            } else {
                double grad_alpha = (f_new - f_x - grad_dot_d * alpha)/(alpha * alpha);
                alpha = cubicInterpolate(alpha_prev, alpha, f_prev, grad_dot_d, 
                                       f_new, grad_alpha);
                
                if (alpha < 0.1 * alpha_prev || alpha > 0.9 * alpha_prev) {
                    alpha = alpha_prev * 0.5;  
                }
            }
        } else {
            alpha = quadraticInterpolate(alpha, 0.0, f_new, grad_dot_d, f_x);
            if (alpha < 0.1 * INITIAL_STEP_SIZE || alpha > 0.9 * INITIAL_STEP_SIZE) {
                alpha = INITIAL_STEP_SIZE * 0.5;
            }
        }
        
        alpha_prev = alpha;
        f_prev = f_new;
    }
    if (alpha < 1e-4) {  
        return 0.5;     // Prevent excessively small steps
    }
    return alpha; 
}


// Safe cubic interpolation with bounds and numerical checks
double safeCubicInterpolate(double alpha0, double alpha1, double phi0, double dphi0, double phi1, double dphi1) {
    if (alpha0 > alpha1) {
        std::swap(alpha0, alpha1);
        std::swap(phi0, phi1);
        std::swap(dphi0, dphi1);
    }
    
    double d1;
    try {
        d1 = dphi0 + dphi1 - 3*(phi1 - phi0)/(alpha1 - alpha0);
    } catch (...) {
        return 0.5 * (alpha0 + alpha1);  
    }
    
    if (std::isnan(d1) || std::isinf(d1)) {
        return 0.5 * (alpha0 + alpha1);
    }
    
    double discriminant = d1*d1 - dphi0*dphi1;
    
    if (discriminant < 0) {
        return 0.5 * (alpha0 + alpha1);
    }
    
    double d2;
    try {
        d2 = std::copysign(sqrt(discriminant), alpha1 - alpha0);
    } catch (...) {
        return 0.5 * (alpha0 + alpha1);  
    }
    
    double denominator = dphi0 - dphi1 + 2*d2;
    
    if (std::abs(denominator) < 1e-10) {
        return 0.5 * (alpha0 + alpha1);
    }
    
    double result;
    try {
        result = alpha0 + (alpha1 - alpha0)*(dphi0 + d2 - d1)/denominator;
    } catch (...) {
        return 0.5 * (alpha0 + alpha1);  
    }
    
    if (std::isnan(result) || std::isinf(result)) {
        return 0.5 * (alpha0 + alpha1);
    }
    
    return std::max(alpha0 + 0.1 * (alpha1 - alpha0), 
                   std::min(alpha1 - 0.1 * (alpha1 - alpha0), result));
}



double wolfeInterpolationLineSearch(
    const vector<double>& x, 
    const vector<double>& d,
    const function<double(vector<double>)>& f, 
    const function<vector<double>(vector<double>)>& grad, 
    const vector<double>& gradient
) {
    const double f_x = f(x);
    const double grad_dot_d = dotProduct(gradient, d);
    double alpha = INITIAL_STEP_SIZE; 
    const int N = x.size();
    vector<double> x_new(N);
    
    double alpha_lo = 0.0;
    double alpha_hi = numeric_limits<double>::infinity();
    double f_lo = f_x;
    double dphi_lo = grad_dot_d;
    
    for (int iter = 0; iter < 20; ++iter) {  
        for (int i = 0; i < N; ++i) {
            x_new[i] = x[i] + alpha * d[i];
        }
        
        double f_new = f(x_new);
        
        if (f_new > f_x + C1 * alpha * grad_dot_d || (f_new >= f_lo && iter > 0)) { 
            alpha_hi = alpha;
            alpha = safeCubicInterpolate(alpha_lo, alpha_hi, f_lo, dphi_lo, f_new, 
                                   (f_new - f_x - grad_dot_d * alpha)/(alpha * alpha));
            continue;
        }
        
        vector<double> grad_new = grad(x_new);
        double dphi_new = dotProduct(grad_new, d);
        
        if (abs(dphi_new) <= -C2 * grad_dot_d) {  
            return alpha;
        }
        
        if (dphi_new >= 0) {
            alpha_hi = alpha;
            alpha = safeCubicInterpolate(alpha_lo, alpha_hi, f_lo, dphi_lo, f_new, dphi_new);
        } else {
            alpha_lo = alpha;
            f_lo = f_new;
            dphi_lo = dphi_new;
            
            if (alpha_hi == numeric_limits<double>::infinity()) {
                alpha *= 2;
            } else {
                alpha = safeCubicInterpolate(alpha_lo, alpha_hi, f_lo, dphi_lo, f_new, dphi_new);
            }
        }
        
        if (alpha < WOLFE_INTERP_MIN) {  
            return WOLFE_INTERP_MIN;
        }
    }
    
    return alpha;  
}



class CudaStream {
    private:
        hipStream_t stream_;
        
    public:
        CudaStream() { hipStreamCreate(&stream_); }
        ~CudaStream() { hipStreamDestroy(stream_); }
        hipStream_t get() { return stream_; }
    };



vector<double> LBFGS_CUDA(
    const function<double(vector<double>)> f,
    const function<vector<double>(vector<double>)> grad,
    const vector<double> x0,
    const string line_search_method,
    const int max_iterations,
    const int m,
    const double tolerance
) {
    const int size = x0.size();
    cout << "Starting" << endl;

    CudaStream compute_stream;
    CudaStream transfer_stream;

    function<double(const vector<double>&, const vector<double>&, 
        const function<double(vector<double>)>&, const vector<double>&)> line_search;

    if (line_search_method == "backtracking") {
        line_search = backtrackingLineSearch;
    } else if (line_search_method == "interpolation") {
        line_search = armijoInterpolationLineSearch;
    } else if (line_search_method == "wolfe") {
        line_search = [&grad](const vector<double>& x, const vector<double>& d,
                            const function<double(vector<double>)>& f,
                            const vector<double>& gradient) {
            return wolfeInterpolationLineSearch(x, d, f, grad, gradient);
        };
    } else if (line_search_method == "backtracking_wolfe") {
        line_search = [&grad](const vector<double>& x, const vector<double>& d,
                            const function<double(vector<double>)>& f,
                            const vector<double>& gradient) {
            return backtrackingWolfeLineSearch(x, d, f, grad, gradient);
        };
    } else {
        throw invalid_argument("Unknown line search method: " + line_search_method);
    }

    double *d_x, *d_g, *d_d, *d_q, *d_r;
    checkCudaError(hipMalloc(&d_x, size * sizeof(double)), "Failed to allocate memory for d_x");
    checkCudaError(hipMalloc(&d_g, size * sizeof(double)), "Failed to allocate memory for d_g");
    checkCudaError(hipMalloc(&d_d, size * sizeof(double)), "Failed to allocate memory for d_d");
    checkCudaError(hipMalloc(&d_q, size * sizeof(double)), "Failed to allocate memory for d_q");
    checkCudaError(hipMalloc(&d_r, size * sizeof(double)), "Failed to allocate memory for d_r");

    std::vector<double*> s_history(m), y_history(m);
    for (int i = 0; i < m; ++i) {
        checkCudaError(hipMalloc(&s_history[i], size * sizeof(double)), "Failed to allocate memory for s_history");
        checkCudaError(hipMalloc(&y_history[i], size * sizeof(double)), "Failed to allocate memory for y_history");
    }

    double* d_x_new;
    double* d_g_new;
    checkCudaError(hipMalloc(&d_x_new, size * sizeof(double)), "Failed to allocate memory for d_x_new");
    checkCudaError(hipMalloc(&d_g_new, size * sizeof(double)), "Failed to allocate memory for d_g_new");

    std::vector<double> gradient(size, 0.0);

    checkCudaError(hipMemcpyAsync(d_x, x0.data(), size * sizeof(double), 
                                    hipMemcpyHostToDevice, transfer_stream.get()), 
                    "Failed to copy x0 to d_x asynchronously");

    hipblasHandle_t cublasHandle;
    checkCublasError(hipblasCreate(&cublasHandle), "Failed to initialize cuBLAS");

    checkCublasError(hipblasSetStream(cublasHandle, compute_stream.get()), 
                    "Failed to set compute stream for cuBLAS");

    int threadsPerBlock = 256;
    int numBlocks = (size + threadsPerBlock - 1) / threadsPerBlock;

    vector<double> alpha(m);
    vector<double> rho(m);

    checkCudaError(hipStreamSynchronize(transfer_stream.get()), 
                    "Failed to synchronize transfer stream after initial data copy");

    for (int k = 0; k < max_iterations; ++k) {
        if (k == 0) {
            gradient = grad(x0);

            checkCudaError(hipMemcpyAsync(d_g, gradient.data(), size * sizeof(double), 
                                            hipMemcpyHostToDevice, transfer_stream.get()), 
                            "Failed to copy gradient to d_g asynchronously");

            checkCudaError(hipStreamSynchronize(transfer_stream.get()), 
                            "Failed to synchronize transfer stream after gradient copy");

            negateVector<<<numBlocks, threadsPerBlock, 0, compute_stream.get()>>>(d_g, d_d, size);
            
        } else {
            checkCudaError(hipMemcpyAsync(d_q, d_g, size * sizeof(double), 
                                            hipMemcpyDeviceToDevice, compute_stream.get()), 
                            "Failed to copy gradient to q asynchronously");

            for (int i = k - 1; i >= max(0, k - m); --i) {
                double si_yi;
                checkCublasError(hipblasDdot(cublasHandle, size, s_history[i % m], 1, y_history[i % m], 1, &si_yi),
                                "CUBLAS ddot for si_yi failed");
                
                if (si_yi <= 1e-10) continue;  
                
                rho[i % m] = 1.0 / si_yi;
                
                double si_q;
                checkCublasError(hipblasDdot(cublasHandle, size, s_history[i % m], 1, d_q, 1, &si_q),
                                "CUBLAS ddot for si_q failed");
                
                alpha[i % m] = rho[i % m] * si_q;
                double neg_alpha = -alpha[i % m];
                checkCublasError(hipblasDaxpy(cublasHandle, size, &neg_alpha, y_history[i % m], 1, d_q, 1),
                                "CUBLAS daxpy failed");
            }

            if (k > 0) {
                double ys, yy;
                int last = (k - 1) % m;
                checkCublasError(hipblasDdot(cublasHandle, size, s_history[last], 1, y_history[last], 1, &ys),
                                "CUBLAS ddot for ys failed");
                checkCublasError(hipblasDdot(cublasHandle, size, y_history[last], 1, y_history[last], 1, &yy),
                                "CUBLAS ddot for yy failed");
                
                if (yy > 0 && ys > 1e-10) {  
                    double gamma = ys / yy;
                    scaleByRho<<<numBlocks, threadsPerBlock, 0, compute_stream.get()>>>(d_q, d_r, gamma, size);
                } else {
                    double gamma = 1.0;
                    scaleByRho<<<numBlocks, threadsPerBlock, 0, compute_stream.get()>>>(d_q, d_r, gamma, size);
                }
            } else {
                checkCudaError(hipMemcpyAsync(d_r, d_q, size * sizeof(double), 
                                                hipMemcpyDeviceToDevice, compute_stream.get()),
                                "Failed to copy q to r asynchronously");
            }

            for (int i = max(0, k - m); i < k; ++i) {
                double yi_r;
                checkCublasError(hipblasDdot(cublasHandle, size, y_history[i % m], 1, d_r, 1, &yi_r),
                                "CUBLAS ddot for yi_r failed");
                
                double beta = rho[i % m] * yi_r;
                double diff = alpha[i % m] - beta;
                checkCublasError(hipblasDaxpy(cublasHandle, size, &diff, s_history[i % m], 1, d_r, 1),
                                "CUBLAS daxpy failed");
            }

            negateVector<<<numBlocks, threadsPerBlock, 0, compute_stream.get()>>>(d_r, d_d, size);
        }

        checkCudaError(hipStreamSynchronize(compute_stream.get()), 
                        "Failed to synchronize compute stream");

        std::vector<double> x_host(size), d_host(size);
        checkCudaError(hipMemcpyAsync(x_host.data(), d_x, size * sizeof(double), 
                                        hipMemcpyDeviceToHost, transfer_stream.get()),
                        "Failed to copy d_x to host asynchronously");
        checkCudaError(hipMemcpyAsync(d_host.data(), d_d, size * sizeof(double), 
                                        hipMemcpyDeviceToHost, transfer_stream.get()),
                        "Failed to copy d_d to host asynchronously");

        checkCudaError(hipStreamSynchronize(transfer_stream.get()), 
                        "Failed to synchronize transfer stream");

        double step_size = line_search(x_host, d_host, f, gradient);

        if (step_size < 1e-10) {
            cout << "Warning: Line search failed at iteration " << k << endl;
            std::vector<double> x_final(size);
            checkCudaError(hipMemcpyAsync(x_final.data(), d_x, size * sizeof(double), 
                                            hipMemcpyDeviceToHost, transfer_stream.get()),
                        "Failed to copy d_x to host asynchronously");
            checkCudaError(hipStreamSynchronize(transfer_stream.get()), 
                            "Failed to synchronize transfer stream");
            return x_final;
        }

        cout << "alpha: " << step_size << endl; 

        updateSolution<<<numBlocks, threadsPerBlock, 0, compute_stream.get()>>>(
            d_x, d_d, d_x_new, step_size, size);

        checkCudaError(hipStreamSynchronize(compute_stream.get()), 
                        "Failed to synchronize compute stream after solution update");

        std::vector<double> x_new_host(size);
        checkCudaError(hipMemcpyAsync(x_new_host.data(), d_x_new, size * sizeof(double), 
                                        hipMemcpyDeviceToHost, transfer_stream.get()),
                        "Failed to copy d_x_new to host asynchronously");
        
        checkCudaError(hipStreamSynchronize(transfer_stream.get()), 
                        "Failed to synchronize transfer stream");

        std::vector<double> g_new_host = grad(x_new_host);

        checkCudaError(hipMemcpyAsync(d_g_new, g_new_host.data(), size * sizeof(double), 
                                        hipMemcpyHostToDevice, transfer_stream.get()),
                        "Failed to copy updated gradient to d_g_new asynchronously");

        checkCudaError(hipStreamSynchronize(transfer_stream.get()), 
                        "Failed to synchronize transfer stream after gradient update");

        updateVectors<<<numBlocks, threadsPerBlock, 0, compute_stream.get()>>>(
            s_history[k % m], d_x_new, d_x, y_history[k % m], d_g_new, d_g, size);

        checkCudaError(hipMemcpyAsync(d_x, d_x_new, size * sizeof(double), 
                                        hipMemcpyDeviceToDevice, compute_stream.get()),
                        "Failed to copy d_x_new to d_x asynchronously");
        checkCudaError(hipMemcpyAsync(d_g, d_g_new, size * sizeof(double), 
                                        hipMemcpyDeviceToDevice, compute_stream.get()),
                        "Failed to copy d_g_new to d_g asynchronously");

        checkCudaError(hipStreamSynchronize(compute_stream.get()), 
                        "Failed to synchronize compute stream after preparing for next iteration");

        double g_dot_g = 0.0;
        checkCublasError(hipblasDdot(cublasHandle, size, d_g, 1, d_g, 1, &g_dot_g),
                        "CUBLAS ddot for g failed");
        double norm_g = std::sqrt(g_dot_g);

        cout << "Iteration " << k << ": norm_g = " << norm_g << endl;
        cout << "Optimum value: " << f(x_new_host) << endl;
        
        if (norm_g <= tolerance) {
            cout << "Convergence achieved at iteration " << k << endl;
            break;
        }
    }

    std::vector<double> x_final(size);
    checkCudaError(hipMemcpyAsync(x_final.data(), d_x, size * sizeof(double), 
                                    hipMemcpyDeviceToHost, transfer_stream.get()),
                    "Failed to copy d_x to host asynchronously");
    checkCudaError(hipStreamSynchronize(transfer_stream.get()), 
                    "Failed to synchronize transfer stream after final copy");

    hipFree(d_x);
    hipFree(d_g);
    hipFree(d_d);
    hipFree(d_q);
    hipFree(d_r);
    for (int i = 0; i < m; ++i) {
        hipFree(s_history[i]);
        hipFree(y_history[i]);
    }
    hipFree(d_x_new);
    hipFree(d_g_new);
    hipblasDestroy(cublasHandle);

    return x_final;
}



double quadratic(const vector<double> &X) {
    double sum = 0.0;
    for (const double x : X) {
        sum += (x - 1) * (x - 1);
    }
    return sum;
}

vector<double> quadratic_grad(const vector<double> &X) {
    vector<double> grad(X.size());
    for (size_t i = 0; i < X.size(); i++) {
        grad[i] = 2.0 * (X[i] - 1);
    }
    return grad;
}

double rosenbrock(const vector<double> &X) {
    double sum = 0.0;
    for (size_t i = 0; i < X.size()-1; i++) {
        double term1 = X[i+1] - X[i] * X[i];
        double term2 = 1 - X[i];
        sum += 100.0 * term1 * term1 + term2 * term2; 
    }
    return sum;
}

vector<double> rosenbrock_grad(const vector<double> &X) {
    vector<double> grad(X.size(), 0.0);
    for (size_t i = 0; i < X.size()-1; i++) {
        double term1 = 2.0 * (X[i] - 1);
        double term2 = X[i+1] - X[i] * X[i];
        grad[i] += term1 - 400.0 * X[i] * term2;
        grad[i+1] += 200.0 * term2;
    }
    return grad;
}

int main() {
    unsigned seed = 42;  
    std::mt19937 gen(seed); 
    std::uniform_real_distribution<> dis(-2, 2); 
    std::vector<double> x0(50000);
    for (double& num : x0) {
        num = dis(gen); 
    }

    cout << "First x: ";
    for (double xi : x0) {
        cout << xi << " ";
    }
    std::vector<double> optimum = LBFGS_CUDA(rosenbrock, rosenbrock_grad, x0, "wolfe", 50000, 10, 1e-1);

    cout << "Found solution: ";
    for (double xi : optimum) {
        cout << xi << " ";
    }
    cout << endl;
    cout << "Optimum value: " << rosenbrock(optimum) << endl;

    return 0;
}
